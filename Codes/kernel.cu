#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gdal.h"
#include "gdal_priv.h"
#include "gdalwarper.h"
#include <stdio.h>
#include<iostream>
#include"trans.h"
#define num_thread 256
#define num_block 32
__device__ float ftest(int i,int j)
{
	 const float f[]={161,18.51,10.13,7.71,6.61,5.99,5.59,5.32,5.12,4.96,
		4.84,4.75,4.67,4.60,4.54,4.49,4.45,4.41,4.38,4.35,
		4.32,4.30,4.28,4.26,4.24,4.22,4.21,4.20,4.18,4.17,
		4.16,4.15,4.14,4.13,4.121,4.113,4.105,4.098,4.091,4.085,
		4.079,4.073,4.07,4.06,4.06,4.05,4.05,4.043,4.038,4.034
	};
	return f[j];
}
float Stddve(float **a,int n,int width,int height)
{
	float stddve=0,sumx=0,sumxx=0;
	for(int i=0;i<width*height;i++)
	{
		sumx+=a[n][i];
		sumxx+=a[n][i]*a[n][i];
	}
	stddve=sqrt(sumxx/(width*height)-(sumx/(width*height))*(sumx/(width*height)));
	return stddve;
}
__global__ void limit_a_CalcuRela_pairs(float **image_pairs,int num_pairs, int Height,int Width, int Win_size1,float M_err,int BandNum,int current,int *location_p,float *r,float *threshold_d,int task_height  )
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	const int Idx=num_thread*bid+tid;
	int b=Idx/(Height*Width);
	int j=(Idx-(Height*Width)*b)/Width;
	int i=(Idx-(Height*Width)*b)%Width;
	int k_h=0;
	float dx,dy;
	float sumx,sumy,sumxy,sumxx,sumyy;
	int num=0;
	int ii=0;
	for(int kkk=Idx;kkk<Height*Width;kkk=kkk+num_thread*num_block)
	{
		b=kkk/(Height*Width);
		j=(kkk-(Height*Width)*b)/Width;
		i=(kkk-(Height*Width)*b)%Width;
		dx=0,dy=0;
		num=0;
		sumyy=0;
		sumx=0;
		sumy=0;
		sumxy=0;
		sumxx=0;
		for(int ii=0;ii<BandNum-1;ii++)
		{
			if(image_pairs[ii][j*Width+i]==image_pairs[ii+1][j*Width+i]&&image_pairs[ii+BandNum*num_pairs][j*Width+i]==image_pairs[ii+BandNum*num_pairs+1][j*Width+i])
				num++;
		}

		if(num!=(BandNum-1)||(BandNum==1))
		{
			for( ii=0;ii<BandNum;ii++)
			{
				for(k_h=0;k_h<num_pairs;k_h++)
				{
				sumxy=sumxy+image_pairs[ii+k_h*BandNum][j*Width+i]*image_pairs[ii+(num_pairs+k_h)*BandNum][j*Width+i];
				sumx=sumx+image_pairs[ii+k_h*BandNum][j*Width+i];
				sumy=sumy+image_pairs[ii+(num_pairs+k_h)*BandNum][j*Width+i];
				sumxx=sumxx+image_pairs[ii+k_h*BandNum][j*Width+i]*image_pairs[ii+k_h*BandNum][j*Width+i];
				sumyy=sumyy+image_pairs[ii+(num_pairs+k_h)*BandNum][j*Width+i]*image_pairs[ii+(num_pairs+k_h)*BandNum][j*Width+i];
				}

			}
			dx=sqrt(sumxx/(BandNum*num_pairs)-(sumx/(BandNum*num_pairs))*(sumx/(BandNum*num_pairs)));
			dy=sqrt(sumyy/(BandNum*num_pairs)-(sumy/(BandNum*num_pairs))*(sumy/(BandNum*num_pairs)));
			r[j*Width+i]=(sumxy/(BandNum*num_pairs)-sumx*sumy/(BandNum*BandNum*num_pairs*num_pairs))/(dx*dy);
			if(BandNum==1&&r[j*Width+i]>0)
		      r[j*Width+i]=1;
			if(BandNum==1&&r[j*Width+i]<0)
		      r[j*Width+i]=-1;
		}
		else
		{
			r[j*Width+i]=1;
		}
		if(r[j*Width+i]!=r[j*Width+i])
			r[j*Width+i]=0;
	}
}
__global__ void Blending2_pairs(float **image_pairs,int num_pairs,int Height,int Width, int Win_size1,float M_err,int BandNum,int current,int *location_p,float *r,float *threshold_d,int task_height,float _nodata)
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	const int Idx=num_thread*bid+tid;
	int b=Idx/(Height*Width);
	int j=(Idx-(Height*Width)*b)/Width;
	int i=(Idx-(Height*Width)*b)%Width;
	int rmin,rmax,smin,smax;
	int r1,s1;
	int Result1=0,m=0;
	int n1;
	float dy;
	float sum,weight_all,weight;
	int k_h=0,k_p=0;
	float aa=0;
	float pix_sum1,pix_sum2;
	//double Aver11;
	//double Aver22;
	float Aver[8];
	float Average1[80],Average3[80];
	float d=0,wi=0;
	float sumx,sumy,sumxy,sumxx,sumyy;
	//float T_weight[8];
	float Aver_all;
	for(int kkk=Idx+current*Width;kkk<(current+task_height)*Width;kkk=kkk+num_thread*num_block)
	{
	
		aa=0;
		pix_sum1=0;
		pix_sum2=0;
		for(m=0;m<80;m++)
		{
			Average1[m]=0;
			Average3[m]=0;
		}
		b=kkk/(Height*Width);
		j=(kkk-(Height*Width)*b)/Width;
		i=(kkk-(Height*Width)*b)%Width;
		for(m=0;m<BandNum;m++)
		{
			pix_sum1+=image_pairs[m][i+Width*j];
			pix_sum2+=image_pairs[m+BandNum][i+Width*j];
		}
		if(fabs(pix_sum1-_nodata)>1e-6&&fabs(pix_sum2-_nodata)>1e-6)
		{
			n1=0;
			weight_all=0,weight=0;
			sum=0;
			sumx=0;
			sumy=0;
		/*	for(m=0;m<8;m++)
			{
				Aver[m]=0;
			}*/
			sumxy=0;
			sumxx=0;
			sumyy=0;
			if(i-Win_size1/2<=0)
				rmin=0;
			else
				rmin = i-Win_size1/2;

			if(i+Win_size1/2>=Width-1)
				rmax = Width-1;
			else
				rmax = i+Win_size1/2;

			if(j-Win_size1/2<=0)
				smin=0;
			else
				smin = j-Win_size1/2;

			if(j+Win_size1/2>=Height-1)
				smax = Height-1;
			else
				smax = j+Win_size1/2;
			r1=rmin,s1=smin;
			for(r1=rmin;r1<=rmax;r1++)
			{
				for(s1=smin;s1<=smax;s1++)
				{  

					Result1=0;	
					for( m=0;m<BandNum*num_pairs;m++)
					{  
						if(fabs(image_pairs[m][ r1+Width*s1]-image_pairs[m][ i+Width*j])<=threshold_d[m])//??
						{
							Result1++;
						}
						else
							break;
					}	

					if(Result1==BandNum*num_pairs )
					{	
						location_p[n1+Idx*Win_size1*Win_size1]=r1+Width*s1;
						d=1+sqrt((float)((r1-i)*(r1-i)+(s1-j)*(s1-j)))/(float)(Win_size1/2);
						weight=1.0/((1.0-r[r1+Width*s1])*d+0.0000001);
						for( m=0;m<BandNum*num_pairs;m++)
						{
<<<<<<< HEAD
							Average1[m]+=(image_pairs[m%BandNum +2*num_pairs*BandNum][r1+Width*s1]-image_pairs[m+num_pairs*BandNum][r1+Width*s1])*weight;
=======
							Average1[m]+=(image_pairs[m%BandNum+2*num_pairs*BandNum][r1+Width*s1]-image_pairs[m+num_pairs*BandNum][r1+Width*s1])*weight;
>>>>>>> 377e39ad22c81bd8790d7be38913158d7e290c00
							//Average2[m]+=(BufferIn55[m][r1+Width*s1]-BufferIn44[m][r1+Width*s1])*weight;
							Average3[m]+=image_pairs[m][r1+Width*s1]*weight;
							//Average4[m]+=BufferIn33[m][r1+Width*s1]*weight;
						}
						weight_all+=weight;
						n1++;
					}
				}
			}

			if(n1>5)
			{
				for(m=0;m<BandNum;m++)
				{
					sumx=0;
					sumy=0;
					sumxy=0;
					sumxx=0;
					sumyy=0;
					for(k_p=0;k_p<8;k_p++)
					{
						Aver[k_p]=0;
					}
						Aver_all=0;
					for(k_h=0;k_h<n1;k_h++)
					{
						for(k_p=0;k_p<num_pairs;k_p++)
						{
						sumxy=sumxy+image_pairs[m+k_p*BandNum][location_p[k_h+Idx*Win_size1*Win_size1]]*image_pairs[m+(k_p+num_pairs)*BandNum][location_p[k_h+Idx*Win_size1*Win_size1]];
						sumx=sumx+image_pairs[m+k_p*BandNum][location_p[k_h+Idx*Win_size1*Win_size1]];
						sumy=sumy+image_pairs[m+(k_p+num_pairs)*BandNum][location_p[k_h+Idx*Win_size1*Win_size1]];
						sumxx=sumxx+image_pairs[m+k_p*BandNum][location_p[k_h+Idx*Win_size1*Win_size1]]*image_pairs[m+k_p*BandNum][location_p[k_h+Idx*Win_size1*Win_size1]];
						sumyy=sumyy+image_pairs[m+(k_p+num_pairs)*BandNum][location_p[k_h+Idx*Win_size1*Win_size1]]*image_pairs[m+(k_p+num_pairs)*BandNum][location_p[k_h+Idx*Win_size1*Win_size1]];
						}
					}
					dy=sqrt(sumyy/(n1*num_pairs)-(sumy/(n1*num_pairs))*(sumy/(n1*num_pairs)));
					if(dy>M_err)
					{
						aa=(sumxy-sumx*sumy/(num_pairs*n1))/(sumyy-sumy*sumy/(n1*num_pairs));
						if(aa>5||aa<0)
						{
							aa=1;
						}
					}
					else
					{
						aa=1.0;
					}
					for( r1=rmin;r1<=rmax;r1++)
					{
						for( s1=smin;s1<=smax;s1++)
						{  
							for(k_p=0;k_p<num_pairs;k_p++)
							{
								Aver[k_p]+=image_pairs[m+2*num_pairs*BandNum][r1+Width*s1]-image_pairs[m+(num_pairs+k_p)*BandNum][r1+Width*s1];
								//Aver22+=BufferIn55[m][r1+Width*s1]-BufferIn44[m][r1+Width*s1];
							}
						}
					}
					for(k_p=0;k_p<num_pairs;k_p++)
					{
						Aver[k_p]=fabs(Aver[k_p])/((float)((rmax-rmin+1)*(smax-smin+1)))+0.0000000001;
						//Aver22+=BufferIn55[m][r1+Width*s1]-BufferIn44[m][r1+Width*s1];
					}
					for(k_p=0;k_p<num_pairs;k_p++)
					{
						Aver_all+=1.0/Aver[k_p];
						//Aver22+=BufferIn55[m][r1+Width*s1]-BufferIn44[m][r1+Width*s1];
					}
					for(k_p=0;k_p<num_pairs;k_p++)
					{
						Aver[k_p]=(1.0/Aver[k_p])/Aver_all;
						//Aver22+=BufferIn55[m][r1+Width*s1]-BufferIn44[m][r1+Width*s1];
					}
					image_pairs[m+(2*num_pairs+1)*BandNum][j*Width+i]=0;
					for(k_p=0;k_p<num_pairs;k_p++)
					{
					image_pairs[m+(2*num_pairs+1)*BandNum][j*Width+i]+=(image_pairs[m+k_p*BandNum][j*Width+i]+aa*Average1[m+k_p*BandNum]/weight_all)*Aver[k_p];
					}
					if(image_pairs[m+(2*num_pairs+1)*BandNum][j*Width+i]<0||image_pairs[m+(2*num_pairs+1)*BandNum][j*Width+i]>1)
					{
						image_pairs[m+(2*num_pairs+1)*BandNum][j*Width+i]=0;
						for(k_p=0;k_p<num_pairs;k_p++)
						{
							image_pairs[m+(2*num_pairs+1)*BandNum][j*Width+i]+=Average3[m+k_p*BandNum]*Aver[k_p]/weight_all;
						}
					}
				}
			}
			else
			{
				for(m=0;m<BandNum;m++)
				{
					for(k_p=0;k_p<8;k_p++)
					{
						Aver[k_p]=0;
					}
					Aver_all=0;
					for( r1=rmin;r1<=rmax;r1++)
					{
						for( s1=smin;s1<=smax;s1++)
						{  
							for(k_p=0;k_p<num_pairs;k_p++)
							{
								Aver[k_p]+=image_pairs[m+2*num_pairs*BandNum][r1+Width*s1]-image_pairs[m+(num_pairs+k_p)*BandNum][r1+Width*s1];
								//Aver22+=BufferIn55[m][r1+Width*s1]-BufferIn44[m][r1+Width*s1];
							}
						}
					}
					for(k_p=0;k_p<num_pairs;k_p++)
					{
						Aver[k_p]=fabs(Aver[k_p])/((float)((rmax-rmin+1)*(smax-smin+1)))+0.0000000001;
						//Aver22+=BufferIn55[m][r1+Width*s1]-BufferIn44[m][r1+Width*s1];
					}
					for(k_p=0;k_p<num_pairs;k_p++)
					{
						Aver_all+=1.0/Aver[k_p];
						//Aver22+=BufferIn55[m][r1+Width*s1]-BufferIn44[m][r1+Width*s1];
					}
					for(k_p=0;k_p<num_pairs;k_p++)
					{
						Aver[k_p]=(1.0/Aver[k_p])/Aver_all;
						//Aver22+=BufferIn55[m][r1+Width*s1]-BufferIn44[m][r1+Width*s1];
					}	
					image_pairs[m+(2*num_pairs+1)*BandNum][j*Width+i]=0;
					for(k_p=0;k_p<num_pairs;k_p++)
					{
						image_pairs[m+(2*num_pairs+1)*BandNum][j*Width+i]+=image_pairs[m+k_p*BandNum][j*Width+i]*Aver[k_p];
					}
				}
			}
		}
		else
		{
			for(m=0;m<BandNum;m++)
			{
				image_pairs[m+(2*num_pairs+1)*BandNum][j*Width+i]=0;
			}
		}
	}
}
__global__ void limit_a_CalcuRela(float **BufferIn11,float **BufferIn22,float **BufferIn33,float **BufferIn44,float **BufferIn55, int Height,int Width, int Win_size1,float M_err,int BandNum,int current,int *location_p,float *r,float *threshold_d,int task_height,float _nodata  )
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	const int Idx=num_thread*bid+tid;
	int b=Idx/(Height*Width);
	int j=(Idx-(Height*Width)*b)/Width;
	int i=(Idx-(Height*Width)*b)%Width;
	int r1,s1;
	int k_h=0;
	float dx,dy;
	float sumx,sumy,sumxy,sumxx,sumyy;
	int num=0;
	for(int kkk=Idx;kkk<Height*Width;kkk=kkk+num_thread*num_block)
	{
		b=kkk/(Height*Width);
		j=(kkk-(Height*Width)*b)/Width;
		i=(kkk-(Height*Width)*b)%Width;
		dx=0,dy=0;
		num=0;
		sumyy=0;
		sumx=0;
		sumy=0;
		sumxy=0;
		sumxx=0;
		for(int ii=0;ii<BandNum-1;ii++)
		{
			if(BufferIn11[ii][j*Width+i]==BufferIn11[ii+1][j*Width+i]&&BufferIn22[ii][j*Width+i]==BufferIn22[ii+1][j*Width+i])
				num++;
		}

		if(num!=(BandNum-1)||(BandNum==1))
		{
			for(int ii=0;ii<BandNum;ii++)
			{
				sumxy=sumxy+BufferIn11[ii][j*Width+i]*BufferIn22[ii][j*Width+i]+BufferIn33[ii][j*Width+i]*BufferIn44[ii][j*Width+i];
				sumx=sumx+BufferIn11[ii][j*Width+i]+BufferIn33[ii][j*Width+i];
				sumy=sumy+BufferIn22[ii][j*Width+i]+BufferIn44[ii][j*Width+i];
				sumxx=sumxx+BufferIn11[ii][j*Width+i]*BufferIn11[ii][j*Width+i]+BufferIn33[ii][j*Width+i]*BufferIn33[ii][j*Width+i];
				sumyy=sumyy+BufferIn22[ii][j*Width+i]*BufferIn22[ii][j*Width+i]+BufferIn44[ii][j*Width+i]*BufferIn44[ii][j*Width+i];

			}
			dx=sqrt(sumxx/(BandNum*2)-(sumx/(BandNum*2))*(sumx/(BandNum*2)));
			dy=sqrt(sumyy/(BandNum*2)-(sumy/(BandNum*2))*(sumy/(BandNum*2)));
			r[j*Width+i]=(sumxy/(BandNum*2)-sumx*sumy/(BandNum*BandNum*4))/(dx*dy);
			if(BandNum==1&&r[j*Width+i]>0)
		      r[j*Width+i]=1;
			if(BandNum==1&&r[j*Width+i]<0)
		      r[j*Width+i]=-1;
		}
		else
		{
			r[j*Width+i]=1;
		}
		if(r[j*Width+i]!=r[j*Width+i])
			r[j*Width+i]=0;
	}

	
}
__global__ void Blending2(float **BufferIn11,float **BufferIn22,float **BufferIn33,float **BufferIn44,float **BufferIn55, float **BufferOut,int Height,int Width, int Win_size1,float M_err,int BandNum,int current,int *location_p,float *r,float *threshold_d,int task_height ,float _nodata)
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	const int Idx=num_thread*bid+tid;
	int b=Idx/(Height*Width);
	int j=(Idx-(Height*Width)*b)/Width;
	int i=(Idx-(Height*Width)*b)%Width;
	int rmin,rmax,smin,smax;
	int r1,s1;
	int Result1=0,m=0;
	int n1;
	float dy;
	float sum,weight_all,weight;
	int k_h=0;
	float aa=0;
	float pix_sum1,pix_sum2;
	double Aver11;
	double Aver22;
	float Average1[10],Average2[10],Average3[10],Average4[10];
	float d=0,wi=0;
	float sumx,sumy,sumxy,sumxx,sumyy;
	float T1_weight;
	float T2_weight;
	for(int kkk=Idx+current*Width;kkk<(current+task_height)*Width;kkk=kkk+num_thread*num_block)
	{
		aa=0;
		pix_sum1=0;
		pix_sum2=0;
		for(m=0;m<10;m++)
		{
			Average1[m]=0;
			Average2[m]=0;
			Average3[m]=0;
			Average4[m]=0;
		}
		b=kkk/(Height*Width);
		j=(kkk-(Height*Width)*b)/Width;
		i=(kkk-(Height*Width)*b)%Width;
		for(m=0;m<BandNum;m++)
		{
			pix_sum1+=BufferIn11[m][i+Width*j];
			pix_sum2+=BufferIn33[m][i+Width*j];
		}
		if(fabs(pix_sum1-_nodata)>1e-6&&fabs(pix_sum2-_nodata)>1e-6) 
		{
			n1=0;
			weight_all=0,weight=0;
			sum=0;
			sumx=0;
			Aver11=0;
			Aver22=0;
			sumy=0;
			sumxy=0;
			sumxx=0;
			sumyy=0;
			if(i-Win_size1/2<=0)
				rmin=0;
			else
				rmin = i-Win_size1/2;

			if(i+Win_size1/2>=Width-1)
				rmax = Width-1;
			else
				rmax = i+Win_size1/2;

			if(j-Win_size1/2<=0)
				smin=0;
			else
				smin = j-Win_size1/2;

			if(j+Win_size1/2>=Height-1)
				smax = Height-1;
			else
				smax = j+Win_size1/2;
			r1=rmin,s1=smin;
			for(r1=rmin;r1<=rmax;r1++)
			{
				for(s1=smin;s1<=smax;s1++)
				{  

					Result1=0;	
					for( m=0;m<BandNum;m++)
					{  
						if(fabs(BufferIn11[m][ r1+Width*s1]-BufferIn11[m][ i+Width*j])<=threshold_d[m]&&fabs(BufferIn33[m][ r1+Width*s1]-BufferIn33[m][ i+Width*j])<=threshold_d[m+BandNum])//??
						{
							Result1++;
						}
						else
							break;
					}	

					if(Result1==BandNum )
					{	
						location_p[n1+Idx*Win_size1*Win_size1]=r1+Width*s1;
						d=1+sqrt((float)((r1-i)*(r1-i)+(s1-j)*(s1-j)))/(float)(Win_size1/2);
<<<<<<< HEAD
						weight=1.0/((1.0-r[r1+Width*s1])*d+0.0000001 );
=======
						weight=1.0/((1.0-r[r1+Width*s1])*d+0.0000001);
>>>>>>> 377e39ad22c81bd8790d7be38913158d7e290c00
						for( m=0;m<BandNum;m++)
						{
							Average1[m]+=(BufferIn55[m][r1+Width*s1]-BufferIn22[m][r1+Width*s1])*weight;
							Average2[m]+=(BufferIn55[m][r1+Width*s1]-BufferIn44[m][r1+Width*s1])*weight;
							Average3[m]+=BufferIn11[m][r1+Width*s1]*weight;
							Average4[m]+=BufferIn33[m][r1+Width*s1]*weight;
						}
						weight_all+=weight;
						n1++;
					}
				}
			}

			if(n1>5)
			{
				for(m=0;m<BandNum;m++)
				{
					sumx=0;
					sumy=0;
					sumxy=0;
					sumxx=0;
					sumyy=0;
					Aver11=0;
					Aver22=0;
					for(k_h=0;k_h<n1;k_h++)
					{
						sumxy=sumxy+BufferIn11[m][location_p[k_h+Idx*Win_size1*Win_size1]]*BufferIn22[m][location_p[k_h+Idx*Win_size1*Win_size1]]+BufferIn33[m][location_p[k_h+Idx*Win_size1*Win_size1]]*BufferIn44[m][location_p[k_h+Idx*Win_size1*Win_size1]];
						sumx=sumx+BufferIn11[m][location_p[k_h+Idx*Win_size1*Win_size1]]+BufferIn33[m][location_p[k_h+Idx*Win_size1*Win_size1]];
						sumy=sumy+BufferIn22[m][location_p[k_h+Idx*Win_size1*Win_size1]]+BufferIn44[m][location_p[k_h+Idx*Win_size1*Win_size1]];
						sumxx=sumxx+BufferIn11[m][location_p[k_h+Idx*Win_size1*Win_size1]]*BufferIn11[m][location_p[k_h+Idx*Win_size1*Win_size1]]+BufferIn33[m][location_p[k_h+Idx*Win_size1*Win_size1]]*BufferIn33[m][location_p[k_h+Idx*Win_size1*Win_size1]];
						sumyy=sumyy+BufferIn22[m][location_p[k_h+Idx*Win_size1*Win_size1]]*BufferIn22[m][location_p[k_h+Idx*Win_size1*Win_size1]]+BufferIn44[m][location_p[k_h+Idx*Win_size1*Win_size1]]*BufferIn44[m][location_p[k_h+Idx*Win_size1*Win_size1]];
					}
					dy=sqrt(sumyy/(n1*2)-(sumy/(n1*2))*(sumy/(n1*2)));
					if(dy>M_err)
					{
						aa=(sumxy-sumx*sumy/(2*n1))/(sumyy-sumy*sumy/(n1*2));
						if(aa>5||aa<0)
						{
							aa=1;
						}
					}
					else
					{
						aa=1.0;
					}
					for(int r1=rmin;r1<=rmax;r1++)
					{
						for(int s1=smin;s1<=smax;s1++)
						{  
							Aver11+=BufferIn55[m][r1+Width*s1]-BufferIn22[m][r1+Width*s1];
							Aver22+=BufferIn55[m][r1+Width*s1]-BufferIn44[m][r1+Width*s1];	
						}
					}
					Aver11=fabs(Aver11)/((float)((rmax-rmin+1)*(smax-smin+1)))+0.0000000001;
					Aver22=fabs(Aver22)/((float)((rmax-rmin+1)*(smax-smin+1)))+0.0000000001;
					T1_weight=1.0/Aver11/(1.0/Aver11+1.0/Aver22);
					T2_weight=1.0/Aver22/(1.0/Aver11+1.0/Aver22);	
					BufferOut[m][j*Width+i]=(BufferIn11[m][j*Width+i]+aa*Average1[m]/weight_all)*T1_weight+(BufferIn33[m][j*Width+i]+aa*Average2[m]/weight_all)*T2_weight;
					if(BufferOut[m][j*Width+i]<0)
						BufferOut[m][j*Width+i]=Average3[m]*T1_weight/weight_all+Average4[m]*T2_weight/weight_all;
				}
			}
			else
			{
				for(m=0;m<BandNum;m++)
				{
					Aver11=0;
					Aver22=0;
					for(int r1=rmin;r1<=rmax;r1++)
					{
						for(int s1=smin;s1<=smax;s1++)
						{  
							Aver11+=BufferIn55[m][r1+Width*s1]-BufferIn22[m][r1+Width*s1];
							Aver22+=BufferIn55[m][r1+Width*s1]-BufferIn44[m][r1+Width*s1];		
						}
					}
					Aver11=fabs(Aver11)/((float)((rmax-rmin+1)*(smax-smin+1)))+0.0000000001;
					Aver22=fabs(Aver22)/((float)((rmax-rmin+1)*(smax-smin+1)))+0.0000000001;
					T1_weight=1/Aver11/(1/Aver11+1/Aver22);
					T2_weight=1/Aver22/(1/Aver11+1/Aver22);	
					BufferOut[m][j*Width+i]=BufferIn11[m][j*Width+i]*T1_weight+BufferIn33[m][j*Width+i]*T2_weight;
				}
			}
		}
		else
		{
			for(m=0;m<BandNum;m++)
			{
				BufferOut[m][j*Width+i]=0;
			}
		}
	}
}
void runtest1_pairs(float **sub_area,int Height,int Width,PARAMETER *par,int BandNum,float *std,int current,int task_height)
 {
	 float **dev_sub_area;
	 float **a;
	 float *dev_std,*r;
	 int num_pairs=par->NUM_PAIRS;
	 int windows=par->WIN_SIZE;
	 float _nodata=par->_nodata;
	 float M_err=par->uncertain;
	 a = (float**)malloc(2*(par->NUM_PAIRS+1)*BandNum*sizeof(float*));
	 for(int b=0;b<2*(par->NUM_PAIRS+1)*BandNum;b++)
	 {
		 hipMalloc((void**)&a[b],Height*Width*sizeof(float));
	 }
	 int *Location_P;
	hipMalloc((void***)&dev_sub_area,sizeof(float*)*2*(par->NUM_PAIRS+1)*BandNum);
	hipMalloc((void**)&Location_P,sizeof(float)*par->WIN_SIZE*par->WIN_SIZE*num_block*num_thread);
	hipMalloc((void**)&r,sizeof(float)*Height*Width);
	hipMalloc((void**)&dev_std,sizeof(float)*BandNum*par->NUM_PAIRS);
	 for(int b=0;b<2*(par->NUM_PAIRS+1)*BandNum;b++)
	 {
        hipMemcpy(a[b], sub_area[b],Height*Width*sizeof(float),hipMemcpyHostToDevice);
	 }
	 hipMemcpy(dev_std, std,sizeof(float)*BandNum*par->NUM_PAIRS,hipMemcpyHostToDevice);
	 hipMemcpy(dev_sub_area,a,sizeof(float*)*2*(par->NUM_PAIRS+1)*BandNum,hipMemcpyHostToDevice);
	 limit_a_CalcuRela_pairs<<<num_block, num_thread>>>(dev_sub_area, num_pairs,Height, Width,windows,M_err,BandNum,current,Location_P,r,dev_std,task_height);
	// hipMemGetInfo(&ff, &tt);
	Blending2_pairs<<<num_block, num_thread>>>(dev_sub_area,num_pairs, Height, Width,  windows, M_err,BandNum,current,Location_P,r,dev_std,task_height,_nodata);
	for(int g=0;g<BandNum;g++)
	{
		hipMemcpy(sub_area[g+(2*num_pairs+1)*BandNum],a[g+(2*num_pairs+1)*BandNum],Height*Width*sizeof(float),hipMemcpyDeviceToHost);
	}
	for(int g=0;g<BandNum;g++)
	{
		hipFree(a[g]);
	}
	hipFree(Location_P);
	hipFree(r);
	hipFree(dev_std);
 }
void runtest1(float **BufferIn11,float **BufferIn22,float **BufferIn33,float **BufferIn44,float **BufferIn55,float **BufferOut,int Height,int Width,int Win_size1,float M_err,int BandNum,float *std,int current,int task_height,float _nodata)
{
	float **dev_BufferIn11,**dev_BufferIn22,**dev_BufferIn33,**dev_BufferIn44,**dev_BufferIn55,**dev_BufferOut;//*Changed_BufferIn11,*Changed_BufferIn33;
	float **a,**f,**c,**d,**e,**out;
	//unsigned int ff, tt
	float *dev_std,*r;
	a = (float**)malloc(BandNum*sizeof(float*));
	f = (float**)malloc(BandNum*sizeof(float*));
	c = (float**)malloc(BandNum*sizeof(float*));
	d = (float**)malloc(BandNum*sizeof(float*));
	e = (float**)malloc(BandNum*sizeof(float*));
	out=(float**)malloc(BandNum*sizeof(float*));
	//test=(float*)malloc(BandNum*Height*Width*sizeof(float));
	for(int b=0;b<BandNum;b++)
	{
		hipMalloc((void**)&a[b],Height*Width*sizeof(float));
		hipMalloc((void**)&f[b],Height*Width*sizeof(float));
		hipMalloc((void**)&c[b],Height*Width*sizeof(float));
		hipMalloc((void**)&d[b],Height*Width*sizeof(float));
		hipMalloc((void**)&e[b],Height*Width*sizeof(float));
		hipMalloc((void**)&out[b],Height*Width*sizeof(float));

	}
	//int num_block= Height* Width*BandNum/num_thread+1;
	int *Location_P;
	hipMalloc((void***)&dev_BufferIn11,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferIn22,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferIn33,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferIn44,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferIn55,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferOut,sizeof(float*)*BandNum);
	hipMalloc((void**)&Location_P,sizeof(float)*Win_size1*Win_size1*num_block*num_thread);
	//hipMalloc((void**)&Changed_BufferIn11,sizeof(float)*Height*Width*BandNum);
	//hipMalloc((void**)&Changed_BufferIn33,sizeof(float)*Height*Width*BandNum);
	hipMalloc((void**)&r,sizeof(float)*Height*Width);
	hipMalloc((void**)&dev_std,sizeof(float)*BandNum*2);
	//hipMalloc((void**)&Location_P,sizeof(float)*100*Width*Height*BandNum);
	for(int g=0;g<BandNum;g++)
	{
		hipMemcpy(a[g], BufferIn11[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(f[g], BufferIn22[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(c[g], BufferIn33[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(d[g], BufferIn44[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(e[g], BufferIn55[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
	}
	hipMemcpy(dev_BufferIn11,a,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferIn22,f,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferIn33, c,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferIn44, d,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferIn55, e,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferOut,out,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_std, std,sizeof(float)*BandNum*2,hipMemcpyHostToDevice);
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	//hipSetDevice(0);
	/*size_t ff,tt;
    hipMemGetInfo(&ff, &tt);
	hipEvent_t start,stop;
	hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);*/
	
	limit_a_CalcuRela<<<num_block, num_thread>>>(dev_BufferIn11,dev_BufferIn22,dev_BufferIn33,dev_BufferIn44,dev_BufferIn55, Height, Width,  Win_size1,M_err,BandNum,current,Location_P,r,dev_std,task_height,_nodata);
	// hipMemGetInfo(&ff, &tt);
	Blending2<<<num_block, num_thread>>>(dev_BufferIn11,dev_BufferIn22,dev_BufferIn33,dev_BufferIn44,dev_BufferIn55,dev_BufferOut, Height, Width,  Win_size1, M_err,BandNum,current,Location_P,r,dev_std,task_height,_nodata);
	/*hipEventRecord(stop,0);
	hipEventSynchronize(stop);
    float costtime=0.0f;
    hipEventElapsedTime(&costtime,start,stop);
	std::cout<<costtime/1000<<"  ";*/
//	 hipMemGetInfo(&ff, &tt);
	for(int g=0;g<BandNum;g++)
	{
		hipMemcpy(BufferOut[g],out[g],Height*Width*sizeof(float),hipMemcpyDeviceToHost);
	}
	//hipMemcpy(test,Changed_BufferIn11,sizeof(float)*BandNum*Height*Width,hipMemcpyDeviceToHost);
	//for(int i=0;i<100;i++)
	//std::cout<<test[i]<<" ";
	/*if(BufferOut[1][1]<0)
		std::cout<<"wrong";*/
	for(int g=0;g<BandNum;g++)
	{
		hipFree(a[g]);
		hipFree(f[g]);
		hipFree(c[g]);
		hipFree(d[g]);
		hipFree(e[g]);
		hipFree(out[g]);
	}
	//hipFree(Changed_BufferIn11);
	//hipFree(Changed_BufferIn33);
	hipFree(Location_P);
	hipFree(r);
	hipFree(dev_std);
}
void runtest(float **BufferIn11,float **BufferIn22,float **BufferIn33,float **BufferIn44,float **BufferIn55,float **BufferOut,int Height,int Width,int Win_size1,float M_err,int num_class,int BandNum,float _nodata)
{
	float *std=new float[BandNum*2];
	for(int i=0;i<BandNum;i++)
	{
		std[i]=Stddve(BufferIn11,i,Width,Height)*2.0/num_class;
		std[i+BandNum]=Stddve(BufferIn33,i,Width,Height)*2.0/num_class;
		std::cout<<std[i]<<"  "<<std[i+BandNum]<<"  ";
	}
	int maxnum;
	size_t ff,tt;
	//hipSetDevice(0);
	hipMemGetInfo(&ff, &tt);
	maxnum=(ff-sizeof(float)*Win_size1*Win_size1*num_block*num_thread*2)/(BandNum*sizeof(float)*8);
	int sub_height=maxnum/Width-Win_size1;
	int kk=0;
	int i,j;
	float **sub_BufferIn11,**sub_BufferIn22,**sub_BufferIn33,**sub_BufferIn44,**sub_BufferIn55,**sub_out;
	for(int heiht_all=0;heiht_all<Height;heiht_all+=sub_height)
	{
		int task_start=kk*sub_height;
		int task_end;
		if((kk+1)*sub_height-Height<=0)
			task_end=(kk+1)*sub_height-1;
		else
			task_end=Height-1; 
		int data_start,data_end;
		if(task_start-Win_size1/2<=0)
			data_start= 0;
		else
			data_start=task_start-Win_size1/2;
		if(task_end+Win_size1/2>=Height-1)
			data_end=Height-1;
		else
			data_end=task_end+Win_size1/2;
		int data_height=data_end-data_start+1;
		sub_BufferIn11=(float**)malloc(BandNum*sizeof(float*));
		sub_BufferIn22=(float**)malloc(BandNum*sizeof(float*));
		sub_BufferIn33=(float**)malloc(BandNum*sizeof(float*));
		sub_BufferIn44=(float**)malloc(BandNum*sizeof(float*));
		sub_BufferIn55=(float**)malloc(BandNum*sizeof(float*));
		sub_out=(float**)malloc(BandNum*sizeof(float*));
		for(int b=0;b<BandNum;b++)
		{
			sub_BufferIn11[b]=new float[data_height*Width];
			sub_BufferIn22[b]=new float[data_height*Width];
			sub_BufferIn33[b]=new float[data_height*Width];
			sub_BufferIn44[b]=new float[data_height*Width];
			sub_BufferIn55[b]=new float[data_height*Width];
			sub_out[b]=new float[data_height*Width];
		}
		int copy;
		for(int k=0;k<BandNum;k++)
		{
			copy=0;
			for( i=data_start;i<=data_end;i++)
			{
				for( j=0;j<Width;j++)
				{
					sub_BufferIn11[k][copy*Width+j]=BufferIn11[k][i*Width+j];
					sub_BufferIn22[k][copy*Width+j]=BufferIn22[k][i*Width+j];
					sub_BufferIn33[k][copy*Width+j]=BufferIn33[k][i*Width+j];
					sub_BufferIn44[k][copy*Width+j]=BufferIn44[k][i*Width+j];
					sub_BufferIn55[k][copy*Width+j]=BufferIn55[k][i*Width+j];
				}
				copy++;
			}
		}
		int current=task_start-data_start;
		int task_height=task_end-task_start+1;
		runtest1(sub_BufferIn11,sub_BufferIn22,sub_BufferIn33,sub_BufferIn44,sub_BufferIn55,sub_out,data_height,Width,Win_size1,M_err, BandNum,std,current,task_height,_nodata);
		
		for(int k=0;k<BandNum;k++)
		{
			current=task_start-data_start;
			for(int i=task_start;i<=task_end;i++)
			{
				for(int j=0;j<Width;j++)
				{
					BufferOut[k][i*Width+j]=sub_out[k][current*Width+j];
				}
				current++;
			}
		}
		for(int g=0;g<BandNum;g++)
	{
		delete sub_BufferIn11[g];
		delete sub_BufferIn22[g];
		delete sub_BufferIn33[g];
		delete sub_BufferIn44[g];
		delete sub_BufferIn55[g];
		delete sub_out[g];
		/*hipFree(dev_BufferIn11[g]);
		hipFree(dev_BufferIn22[g]);
		hipFree(dev_BufferIn33[g]);
		hipFree(dev_BufferIn44[g]);
		hipFree(dev_BufferIn55[g]);
		hipFree(dev_BufferOut[g]);*/
	}
		kk++;
	}
}
void runtest_pairs(CuLayer *psensor,PARAMETER *par,int solve)
{
	int Height=psensor[0].getHeight();
	int Width=psensor[0].getWidth();
	int BandNum=psensor[0].getbandCount();
	float *std=new float[psensor[0].getbandCount()*par->NUM_PAIRS];
	for(int j=0;j<par->NUM_PAIRS;j++)
	{
		for(int i=0;i<psensor[0].getbandCount();i++)
		{

			std[i+psensor[0].getbandCount()*j]=Stddve(psensor[j].getData(),i,psensor[0].getWidth(),psensor[0].getHeight())*2.0/par->class_num;
			std::cout<<std[i+psensor[0].getbandCount()*j]<<"  ";
		}
	}
	int maxnum;
	size_t ff,tt;
	//hipSetDevice(0);
	hipMemGetInfo(&ff, &tt);
	maxnum=(ff-sizeof(float)*par->WIN_SIZE*par->WIN_SIZE*num_block*num_thread*2)/(psensor[0].getbandCount()*sizeof(float)*2*(par->NUM_PAIRS+1))-par->WIN_SIZE;
	int sub_height=maxnum/psensor[0].getWidth();
	int kk=0;
	int i,j,c;
	float **sub_area;
	for(int heiht_all=0;heiht_all<Height;heiht_all+=sub_height)
	{
		int task_start=kk*sub_height;
		int task_end;
		if((kk+1)*sub_height-Height<=0)
			task_end=(kk+1)*sub_height-1;
		else
			task_end=Height-1; 
		int data_start,data_end;
		if(task_start-par->WIN_SIZE/2<=0)
			data_start= 0;
		else
			data_start=task_start-par->WIN_SIZE/2;
		if(task_end+par->WIN_SIZE/2>=Height-1)
			data_end=Height-1;
		else
			data_end=task_end+par->WIN_SIZE/2;
		int data_height=data_end-data_start+1;
		sub_area=(float**)malloc(2*(par->NUM_PAIRS+1)*BandNum*sizeof(float*));
		for(int b=0;b<2*(par->NUM_PAIRS+1)*BandNum;b++)
		{
			sub_area[b]=new float[data_height*Width];
		}
		int copy;
		for(int k=0;k<BandNum;k++)
		{
			copy=0;
			for( i=data_start;i<=data_end;i++)
			{
				for( j=0;j<Width;j++)
				{
					for(c=0;c<2*par->NUM_PAIRS;c++)
					{
						sub_area[k+c*BandNum][copy*Width+j]=psensor[c].getData()[k][i*Width+j];
					}
					sub_area[2*par->NUM_PAIRS*BandNum+k][copy*Width+j]=psensor[2*(par->NUM_PAIRS+solve)].getData()[k][i*Width+j];
				}
				copy++;
			}
		}
		int current=task_start-data_start;
		int task_height=task_end-task_start+1;
		runtest1_pairs(sub_area,data_height,Width,par, BandNum,std,current,task_height);
		
		for(int k=0;k<BandNum;k++)
		{
			current=task_start-data_start;
			for(int i=task_start;i<=task_end;i++)
			{
				for(int j=0;j<Width;j++)
				{
					psensor[2*(par->NUM_PAIRS+solve)+1].getData()[k][i*Width+j]=sub_area[(2*par->NUM_PAIRS+1)*BandNum+k][current*Width+j];
				}
				current++;
			}
		}
		for(int b=0;b<2*(par->NUM_PAIRS+1)*BandNum;b++)
		{
			delete sub_area[b];
			/*hipFree(dev_BufferIn11[g]);
			hipFree(dev_BufferIn22[g]);
			hipFree(dev_BufferIn33[g]);
			hipFree(dev_BufferIn44[g]);
			hipFree(dev_BufferIn55[g]);
			hipFree(dev_BufferOut[g]);*/
		}
		kk++;
	}
}
void Re_fusion3(CuLayer *psensor,PARAMETER *par)
{
	int i,j,m,c;
	long now1 = clock();
	for(c=0;c<par->NUM_PREDICTIONS;c++)
	{
		psensor[2*(par->NUM_PAIRS+c)+1].resize(psensor[0].getWidth(),psensor[0].getHeight(),psensor[0].getbandCount());
		if(par->NUM_PAIRS==2)
		{
			runtest(psensor[0].getData(),psensor[2].getData(),psensor[1].getData(),psensor[3].getData(),psensor[2*(par->NUM_PAIRS+c)].getData(),psensor[2*(par->NUM_PAIRS+c)+1].getData(),psensor[0].getHeight(),psensor[0].getWidth(),par->WIN_SIZE,par->uncertain,par->class_num,psensor[0].getbandCount(),par->_nodata);
			//char* driverName = "GTiff";
			psensor[2*(par->NUM_PAIRS+c)+1].setGeoTransform(psensor[0].getGeoTransform());
			psensor[2*(par->NUM_PAIRS+c)+1].setProjection(psensor[0].getProjection());
			psensor[2*(par->NUM_PAIRS+c)+1].Write(psensor[2*(par->NUM_PAIRS+c)+1].outpath,par->G_Type);
		}
		else
		{
			runtest_pairs(psensor,par,c);
			psensor[2*(par->NUM_PAIRS+c)+1].setGeoTransform(psensor[0].getGeoTransform());
			psensor[2*(par->NUM_PAIRS+c)+1].setProjection(psensor[0].getProjection());
			psensor[2*(par->NUM_PAIRS+c)+1].Write(psensor[2*(par->NUM_PAIRS+c)+1].outpath,par->G_Type);
		}
	}

}
 void Re_fusion2(const char * BufferIn0,const char * BufferIn1,const char * BufferIn2,const char * BufferIn3,const char * BufferIn4,const char * BufferOut,int win_size,int class_num,float M_err,float _nodata)
{
	GDALAllRegister();
	CPLSetConfigOption("GDAL_FILENAME_IS_UTF8","NO"); 
	GDALDataset *Landsat0 = (GDALDataset*) GDALOpen(BufferIn0,GA_ReadOnly);
	int width,height,BandNum;
	width = Landsat0->GetRasterXSize();
	height = Landsat0->GetRasterYSize();
	BandNum = Landsat0->GetRasterCount();
	//height=2000;
   // width=2000;
	float** BufferLandsat_0 = new float*[BandNum];
	int b,k;
	for( b=0;b<BandNum;b++)
	{
		BufferLandsat_0[b] = new float[width*height];
	//	printf("%u\n", BufferLandsat_0[b]);
	}
	
	for( k=0;k<BandNum;k++)
	{
		GDALRasterBand* hInBand1 = Landsat0->GetRasterBand(k+1);
		hInBand1->RasterIO(GF_Read,0,0,width,height,BufferLandsat_0[k],width,height,GDT_Float32,0,0);
	}	

	GDALDataset *MODIS0 = (GDALDataset*) GDALOpen(BufferIn1,GA_ReadOnly);
	float** BufferModis_0 = new float*[BandNum];
	for( b=0;b<BandNum;b++)
	{
		BufferModis_0[b] = new float[width*height];	
	}
	
	for( k=0;k<BandNum;k++)
	{
		GDALRasterBand* hInBand1 = MODIS0->GetRasterBand(k+1);
		hInBand1->RasterIO(GF_Read,0,0,width,height,BufferModis_0[k],width,height,GDT_Float32,0,0);		
	}	

	GDALDataset *Landsat1 = (GDALDataset*) GDALOpen(BufferIn2,GA_ReadOnly);
	float** BufferLandsat_1 = new float*[BandNum];
	for( b=0;b<BandNum;b++)
	{
		BufferLandsat_1[b] = new float[width*height];	
	}
	
	for(k=0;k<BandNum;k++)
	{
		GDALRasterBand* hInBand1 = Landsat1->GetRasterBand(k+1);
		hInBand1->RasterIO(GF_Read,0,0,width,height,BufferLandsat_1[k],width,height,GDT_Float32,0,0);
	}	

	GDALDataset *MODIS1 = (GDALDataset*) GDALOpen(BufferIn3,GA_ReadOnly);
	float** BufferModis_1 = new float*[BandNum];
	for( b=0;b<BandNum;b++)
	{
		BufferModis_1[b] = new float[width*height];	
	}
	
	for( k=0;k<BandNum;k++)
	{
		GDALRasterBand* hInBand1 = MODIS1->GetRasterBand(k+1);
		hInBand1->RasterIO(GF_Read,0,0,width,height,BufferModis_1[k],width,height,GDT_Float32,0,0);		
	}	
	
	GDALDataset *MODIS2 = (GDALDataset*) GDALOpen(BufferIn4,GA_ReadOnly);
	
	float** BufferModis_2 = new float*[BandNum];
	for( b=0;b<BandNum;b++)
	{
		BufferModis_2[b] = new float[width*height];	
	}
	
	for( k=0;k<BandNum;k++)
	{
		GDALRasterBand* hInBand1 = MODIS2->GetRasterBand(k+1);
		hInBand1->RasterIO(GF_Read,0,0,width,height,BufferModis_2[k],width,height,GDT_Float32,0,0);		
		
	}
	
	GDALDataset *LandsatDs;
	char* driverName = "GTiff";
	GDALDriver *pDriver = (GDALDriver*)GDALGetDriverByName(driverName);
	LandsatDs = pDriver->Create(BufferOut,width,height,BandNum,GDT_Float64,NULL);
	double* geos=new double[6];
	Landsat0->GetGeoTransform(geos);
	LandsatDs->SetGeoTransform(geos);
	LandsatDs->SetProjection(Landsat0->GetProjectionRef());
	
	float** BufferOutColor = new float*[BandNum];
	for( b=0;b<BandNum;b++)
	{
		BufferOutColor[b] = new float[width*height];
	}
	//e.Blending2(BufferLandsat_0,BufferModis_0,BufferLandsat_1,BufferModis_1,BufferModis_2,BufferOutColor,height,width,win_size,flag, L_err, M_err, Para_h,BandNum,1.0);
	long now1 = clock();
	 runtest(BufferLandsat_0,BufferModis_0,BufferLandsat_1,BufferModis_1,BufferModis_2,BufferOutColor,height,width,win_size,M_err,BandNum,class_num,_nodata);
	  printf("GPU����ʱ��Ϊ��%dms\n", int(((double)(clock() - now1)) / CLOCKS_PER_SEC * 1000));
	for (b=0;b<BandNum;b++)
	{
		GDALRasterBand* HOut = LandsatDs->GetRasterBand(b+1);
		HOut->RasterIO(GF_Write,0,0,width,height,BufferOutColor[b],width,height,GDT_Float32,0,0);
	}
	GDALClose(Landsat0);
	GDALClose(MODIS0);
	GDALClose(Landsat1);
	GDALClose(MODIS1);
	GDALClose(MODIS2);
	GDALClose(LandsatDs);

	for (b=0;b<BandNum;b++)
	{
		delete []BufferLandsat_0[b];
		delete []BufferModis_0[b];
		delete []BufferLandsat_1[b];
		delete []BufferModis_1[b];
		delete []BufferModis_2[b];
		delete []BufferOutColor[b];
	}
	delete []BufferLandsat_0;
	delete [] BufferModis_0;
	delete []BufferLandsat_1;
	delete [] BufferModis_1;
	delete [] BufferModis_2;
	delete [] BufferOutColor;
}


